#include "hip/hip_runtime.h"
#include <doctest/doctest.h>

#include <astray/api.hpp>

void test()
{
  using scalar_type             = float;
  using vector_type             = ast::vector3<scalar_type>;
  using tableau_type            = ast::dormand_prince_5_tableau<scalar_type>;
  using method_type             = ast::explicit_method<tableau_type>;
  using problem_type            = ast::initial_value_problem<scalar_type, vector_type>;
  using i_controller            = ast::integral_controller                        <scalar_type, tableau_type>;
  using pi_controller           = ast::proportional_integral_controller           <scalar_type, tableau_type>;
  using pid_controller          = ast::proportional_integral_derivative_controller<scalar_type, tableau_type>;
  using i_controller_iterator   = ast::adaptive_step_iterator<method_type, problem_type, i_controller  >;
  using pi_controller_iterator  = ast::adaptive_step_iterator<method_type, problem_type, pi_controller >;
  using pid_controller_iterator = ast::adaptive_step_iterator<method_type, problem_type, pid_controller>;
  
  std::vector<vector_type> input (1);
  std::vector<vector_type> output(1);

  thrust::device_vector<vector_type> device_data = input;
  thrust::for_each(device_data.begin(), device_data.end(), [ ] __device__ (auto& value)
  {
    constexpr auto sigma   = 10.0f;
    constexpr auto rho     = 28.0f;
    constexpr auto beta    = 8.0f / 3.0f;
    const     auto problem = problem_type
    {
      0.0f,                                     /* t0 */
      vector_type(16.0f, 16.0f, 16.0f),         /* y0 */
      [&] __device__ (const float t, const vector_type& y) /* y' = f(t, y) */
      {
        return vector_type(sigma * (y[1] - y[0]), y[0] * (rho - y[2]) - y[1], y[0] * y[1] - beta * y[2]); /* Lorenz system */
      }
    };
    
    auto iterator  = i_controller_iterator   {problem, 1.0f, {}};
    for (auto i = 0; i < 10000; ++i)
      ++iterator;
      
    auto iterator2 = pi_controller_iterator  {problem, 1.0f, {}};
    for (auto i = 0; i < 10000; ++i)
      ++iterator2;
      
    // auto iterator3 = pid_controller_iterator {problem, 1.0f, pid_controller()};
    // for (auto i = 0; i < 10000; ++i)
    //   ++iterator3;

    value = iterator.problem.value;
  });
  thrust::copy(device_data.begin(), device_data.end(), output.begin());
}

TEST_CASE("ast::ode")
{
  test();
}